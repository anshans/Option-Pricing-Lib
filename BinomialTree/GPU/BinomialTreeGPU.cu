#include "hip/hip_runtime.h"
#include "BinomialTreeGPU.hpp"
//Constructors
BinomialTreeGPU::BinomialTreeGPU(float currStockPrice, float u, float d, float r,
                           float strikePrice, float T, int steps) : u(u), d(d), currStockPrice(currStockPrice), steps(steps),
                                                                    T(T), r(r), strikePrice(strikePrice){};
BinomialTreeGPU::BinomialTreeGPU(float currStockPrice, float vol, float r,
                           float strikePrice, float T, int steps) : currStockPrice(currStockPrice), steps(steps),
                                                                    T(T), r(r), strikePrice(strikePrice), u(exp(vol * sqrt(T))), d(-exp(vol * sqrt(T))){};

// Creation of pricce tree
void BinomialTreeGPU::createPriceTree()
{
    //Numer of nodes at beggining of tree
    //layersOfNodes = std::make_unique<std::unique_ptr<PriceNode[]>[]>(steps);
    hipMallocManaged(&layersOfNodes, steps * sizeof(PriceNode **));
    //Create beginning node
    layersOfNodes[0] = new PriceNode();
    layersOfNodes[0][0].stockPrice = currStockPrice;
    //Previous layer(parentLayer) is giving values to next layer (parentLayer+1)
    //Last layer, can't have leaves therefore there is steps - 1.
    fillPriceTree<<<1, 8>>>(layersOfNodes, steps, u, d, &BinomialTreeGPU::numInLayer);
};

__global__ void fillPriceTree(PriceNode **layersOfNodes, int steps, float u, float d, LayerCalculation layerFunction)
{
    //To get better understanding what is going here, check "An Even Easier Introduction to CUDA"
    int index = threadIdx.x;
    int stride = blockDim.x;
    //Allocate memory for nodes and calculate stock prices for each node.
    for (int parentLayer = 0; parentLayer < steps - 1; parentLayer++)
    {
        layersOfNodes[parentLayer + 1] = new PriceNode[layerFunction(parentLayer + 1)];

        for (int nodeInParentLayer = index; nodeInParentLayer < layerFunction(parentLayer); nodeInParentLayer += stride)
        {
            layersOfNodes[parentLayer + 1][nodeInParentLayer].stockPrice = layersOfNodes[parentLayer][nodeInParentLayer].stockPrice * u;
            layersOfNodes[parentLayer + 1][nodeInParentLayer + 1].stockPrice = layersOfNodes[parentLayer][nodeInParentLayer].stockPrice * d;
        }
    }
};

//Calculation of derivative price.
void BinomialTreeGPU::calculateOption(OptionType optionType)
{
    int numOfNodes;
    float p = (exp(r * T) - d) / (u - d);

    for (int layer = steps - 1; layer >= 0; layer--)
    {
        numOfNodes = numInLayer(layer);

        if (layer < steps - 1)
        {
            calculateContractLayer<<<1,8>>>(layersOfNodes, layer, p, strikePrice, numOfNodes, T, r, optionType, BinomialTreeGPU::euContractValue);
        }
        //It's option value at expiration date.
        else
        {
            for (int j = 0; j < numOfNodes; j++)
            {
                float optionValue = layersOfNodes[layer][j].stockPrice - strikePrice;
                layersOfNodes[layer][j].contractPrice = ((optionValue > 0) ? (optionValue) : 0);
            }
        }
    }
};
__global__ void calculateContractLayer(PriceNode **layersOfNodes, int layer, float p, float strikePrice, int numOfNodes, float T, float r, OptionType optionType, ContractCalculation euContractValue)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int j = index; j < numOfNodes; j+=stride)
    {
        float contractValue = euContractValue(layersOfNodes, layer, j, p, T, r);
        if (optionType == OptionType::Eu)
        {
            layersOfNodes[layer][j].contractPrice = contractValue;
        }
        else if (optionType == OptionType::Am)
        {
            layersOfNodes[layer][j].contractPrice = (layersOfNodes[layer][j].stockPrice - strikePrice > contractValue)
                                                        ? layersOfNodes[layer][j].stockPrice - strikePrice
                                                        : contractValue;
        }
    }
}

float BinomialTreeGPU::euContractValue(PriceNode** layersOfNodes, int layer, int j, float p, float T, float r)
{
    //For nodes deeper within the tree, equation 13.2 from Hull book
    //is being used.
    float fu = layersOfNodes[layer + 1][j].contractPrice;
    float fd = layersOfNodes[layer + 1][j + 1].contractPrice;
    //13.2 Hull
    return __exp(-r * T) * (p * fu + (1 - p) * fd);
};

float BinomialTreeGPU::calculateEuropean()
{
    createPriceTree();
    calculateOption(Eu);
    return layersOfNodes[0][0].contractPrice;
};
float BinomialTreeGPU::calculateAmerican()
{
    createPriceTree();
    calculateOption(Am);
    return layersOfNodes[0][0].contractPrice;
};
int BinomialTreeGPU::numInLayer(int layerNum)
{
    //layerNum counts from 0 just like in unique_ptr array!
    return (layerNum + 1);
};